#include "hip/hip_runtime.h"
#include <vector>
#include <cstdlib>
#include <algorithm>
#include <execution>
#include <iostream>
#include <chrono>
#include <tbb/parallel_for.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>

// kernel function to perform vector c = alpha * a + b
// size is the size of the vectors
// __restrict__ tells the compiler a, b, and c will not
// alias to each other (as in a != b, b != c, and a != c
__global__ void saxpy(const float alpha, const float* __restrict__ a, const float* __restrict__ b, float* __restrict__ c, int size); 

int main() {

    // Lets add and scale two vectors together:
    // c = alpha * a + b

    // this is the size of our problem 
    const int size = 1000000;


    // we will create size element vectors with entries of zero
    thrust::host_vector<float> a(size, 0);
    thrust::host_vector<float> b(size, 0);
    thrust::host_vector<float> c(size, 0);

    // std::tranform reads from the range a.cbegin() to a.cend()
    // and will write the result out iteratively to a.begin()
    // it uses the function std::rand to output the result
    std::transform(a.cbegin(), a.cend(), a.begin(), [](auto) { return static_cast<float>(rand()) / RAND_MAX; });
    std::transform(b.cbegin(), b.cend(), b.begin(), [](auto) { return static_cast<float>(rand()) / RAND_MAX; });

    // we set alpha to 1
    float alpha = 1.0f;

    // SEQUENTIAL EXECUTION
    // -------------------------------------------------------------------------------------------------------

    // this gets the current time
    auto start = std::chrono::high_resolution_clock::now();
   
    // we add together using a lambda from a.cbegin() to a.cend()
    // and b.cbegin() until we iterate through a and output to c
    // we use the lambda to capture alpha and add the two numbers
    std::transform(std::execution::seq, a.begin(), a.end(), b.begin(), c.begin(), [alpha](const auto& a, const auto& b) {
        return alpha * a + b;
    });
    
    auto end = std::chrono::high_resolution_clock::now();
    
    // we compute the number of seconds elapsed
    double seq_time = std::chrono::duration<double>(end - start).count();

    // C++ STANDARD PARALLEL EXECUTION
    // -------------------------------------------------------------------------------------------------------

    // we create another vector to output to
    thrust::host_vector<float> c2(size, 0);

    start = std::chrono::high_resolution_clock::now();

    // lets compare to a parallel execution that can be parallelized or vectorized
    // in any way the compiler desires
    // this does the transformation in parallel instead of sequential by specifying par_unseq (parallel unsequenced)
    std::transform(std::execution::par_unseq, a.begin(), a.end(), b.begin(), c2.begin(), [alpha](const auto& a, const auto& b) {
        return alpha * a + b;
    });
    
    end = std::chrono::high_resolution_clock::now();

    double par_time = std::chrono::duration<double>(end - start).count();

    // we double check if we have computed accurately within a 1e-5 tolerance
    bool fail = false;
    for(int i = 0; i < size; ++i) {
        if(std::abs(c[i] - c2[i]) > std::abs(c[i] * 1e-5)) {
            std::cerr << "Error cpp transform: c[" << i << "] do not match " << c[i] << " != " << c2[i] << std::endl;
            std::cerr << "Computed: " << a[i] << " + " << b[i] << std::endl;
            fail = true; 
        }
    }

    if(fail) return 1;

    // INTEL THREAD BUILDING BLOCKS PARALLEL EXECUTION
    // -------------------------------------------------------------------------------------------------------

    start = std::chrono::high_resolution_clock::now();

    // this parallelizes a for loop by asigning each thread of execution a range from r.begin() to r.end()
    // to compute over
    tbb::parallel_for(tbb::blocked_range<int>(0, size), [&](tbb::blocked_range<int> r) {

        for(int i = r.begin(); i < r.end(); ++i) {
            c2[i] = alpha * a[i] + b[i];
        }  

    });
    end = std::chrono::high_resolution_clock::now();
    
    double tbb_time = std::chrono::duration<double>(end - start).count();

    for(int i = 0; i < size; ++i) {
        if(std::abs(c[i] - c2[i]) > std::abs(c[i] * 1e-5)) {
            std::cerr << "Error tbb parallel for: c[" << i << "] do not match " << c[i] << " != " << c2[i] << std::endl;
            std::cerr << "Computed: " << a[i] << " + " << b[i] << std::endl;
            fail = true; 
        }
    }

    // THRUST CUDA GPU PARALLEL EXECUTION
    // -------------------------------------------------------------------------------------------------------

    // we create device vectors which are created in GPU memory
    thrust::device_vector<float> c_kern(size, 0);

    start = std::chrono::high_resolution_clock::now();
   
    // these device vectors are initialized with data from the begining of the corresponding 
    // host (CPU) vector to the end of the vector
    thrust::device_vector<float> a_kern(a.begin(), a.end());
    thrust::device_vector<float> b_kern(b.begin(), b.end());

    // this executes the same transform operation as std::transform but on the GPU or CPU. We mark the lambda with __host__ __device__
    // this enables us to execute this lambda on both the host (CPU) and device (GPU)
    thrust::transform(a_kern.begin(), a_kern.end(), b_kern.begin(), c_kern.begin(), [=] __host__ __device__ (const float& a, const float& b) {
        return alpha * a + b;        
    });

    // we copy the results back from the GPU to the CPU into c2 from the begining to end
    // we start copying from the begining of c_kern
    thrust::copy(c2.begin(), c2.end(), c_kern.begin());
    
    end = std::chrono::high_resolution_clock::now();
    
    double thrust_time = std::chrono::duration<double>(end - start).count();

    for(int i = 0; i < size; ++i) {
        if(std::abs(c[i] - c2[i]) > std::abs(c[i] * 1e-5)) {
            std::cerr << "Error tbb parallel for: c[" << i << "] do not match " << c[i] << " != " << c2[i] << std::endl;
            std::cerr << "Computed: " << a[i] << " + " << b[i] << std::endl;
            fail = true; 
        }
    }

    if(fail) return 1;

    // MANUAL CUDA PARALLEL EXECUTION
    // -------------------------------------------------------------------------------------------------------

    float* c_kern2;
    float* a_kern2;
    float* b_kern2;

    // malloc similar to c malloc but takes pointer to where you write what you allocated.
    // This is allocating the variable a_kern2 of sizeof(float) * size bytes
    // on the GPU.
    hipError_t hipError_t = hipMalloc(&a_kern2, sizeof(float) * size);
    // we must check if an error occured
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }

    hipError_t = hipMalloc(&b_kern2, sizeof(float) * size);
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }

    hipError_t = hipMalloc(&c_kern2, sizeof(float) * size);
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }


    start = std::chrono::high_resolution_clock::now();
  
    // memory copies to gpu similar to C memcpy API plus where to where you are copying 
    hipError_t = hipMemcpy(a_kern2, a.data(), sizeof(float) * size, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }

    hipError_t = hipMemcpy(b_kern2, b.data(), sizeof(float) * size, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }

    // asynchronous launch of GPU with x blocks and y threads per block
    // note that we are setting the number of blocks to (size + 1023) / 1024
    // the number of threads per block is 64 and each block will handle 1024 elements
    // (size + 1023) / 1024 is an approach to calculating the ceiling of size / 1024.0
    // by doing this we can handle cases where 1024 does not evenly divide the size of
    // the vectors in our saxpy
    saxpy<<<(size + 1023) / 1024, 64 >>>(alpha, a_kern2, b_kern2, c_kern2, size);
   
    hipError_t = hipDeviceSynchronize(); // synchronize with GPU
    // this synchronization must occur or we will be unable to accurately get results from
    // the GPU
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }

    // we copy data back into c2 by getting the underlying pointer with c2.data()
    hipError_t = hipMemcpy(c2.data(), c_kern2, sizeof(float) * size, hipMemcpyDeviceToHost);
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }

    end = std::chrono::high_resolution_clock::now();
    
    double cuda_time = std::chrono::duration<double>(end - start).count();

    // we must free our allocated memory on the GPU with hipFree
    hipError_t = hipFree(a_kern2);
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }
    hipError_t = hipFree(b_kern2);
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }
    hipError_t = hipFree(c_kern2);
    if(hipError_t != hipSuccess) {
        std::cerr << "cuda failure" << std::endl;
        return 2;
    }

    for(int i = 0; i < size; ++i) {
        if(std::abs(c[i] - c2[i]) > std::abs(c[i] * 1e-5)) {
            std::cerr << "Error cuda parallel for: c[" << i << "] do not match " << c[i] << " != " << c2[i] << std::endl;
            std::cerr << "Computed: " << a[i] << " + " << b[i] << std::endl;
            fail = true; 
        }
    }

    std::cout << "Duration of parallel cpp version (ms):\t\t" << par_time * 1e3 << std::endl;
    std::cout << "Duration of parallel tbb version (ms):\t\t" << tbb_time * 1e3 << std::endl;
    std::cout << "Duration of parallel thrust version (ms):\t" << thrust_time * 1e3 << std::endl;
    std::cout << "Duration of parallel cuda version (ms):\t\t" << cuda_time * 1e3 << std::endl;
    std::cout << "Duration of sequential version (ms):\t\t" << seq_time * 1e3 << std::endl;
    auto times = std::vector<double>{tbb_time, par_time, thrust_time, cuda_time};
    double best = *std::min_element(times.begin(), times.end());
    std::cout << "Speedup of best:\t\t\t\t" << seq_time / best << std::endl;
    return 0;
}

// kernel function to perform vector c = alpha * a + b
// size is the size of the vectors
// __restrict__ tells the compiler a, b, and c will not
// alias to each other (as in a != b, b != c, and a != c
__global__ void saxpy(const float alpha, const float* __restrict__ a, const float* __restrict__ b, float* __restrict__ c, int size) {

    // each block handles 1024 elements
    // each block has 64 threads
    // each thread in a block handles 4 of those elements
    // we iterate 4 times to handle the 1024 elements
    constexpr int elements_per_thread = 4;
    
    // threadIdx.x is the location within the grid
    // blockIdx.x is the location of the block
    // blockDim.x is the dimension of the block (threads per block)
    // we know blockDim.x = 64 so we substitute 64 in
    int tidx = threadIdx.x; 
    
    // we will block the vector addition with 4 elements per each thread
   
    int bidx = blockIdx.x;

    const float* a_block = a + bidx * 1024;
    const float* b_block = b + bidx * 1024;
    float* c_block = c + bidx * 1024;

    // if we know we are in bounds of the vector or if the vector size is evenly divisible by 1024
    if(bidx < size / 1024 || (size + 1023) / 1024 == size / 1024) {

        // we process 256 floating point calculations per loop
        // float 512 floats loaded and 256 stored
        // this pragma tells the compiler to unroll the loop
        #pragma unroll
        for(int i = 0; i < 1024 / 64 / elements_per_thread; ++i) {
            float4 reg_a = *(reinterpret_cast<const float4*>(a_block) + tidx + i * 64);
            float4 reg_b = *(reinterpret_cast<const float4*>(b_block) + tidx + i * 64);
            
            reg_b.w += alpha * reg_a.w;
            reg_b.x += alpha * reg_a.x;
            reg_b.y += alpha * reg_a.y;
            reg_b.z += alpha * reg_a.z;

            *(reinterpret_cast<float4*>(c_block) + tidx + i * 64) = reg_b;
        }
    } else {
        // this tells the compiler to unroll this loop
        #pragma unroll
        for(int i = 0; i < 1024 / 64 / elements_per_thread; ++i) {

            // if the loads and stores are all in bounds proceed as normal
            if(bidx + tidx + i * 256 + 3 < size) {
                float4 reg_a = *(reinterpret_cast<const float4*>(a_block) + tidx + i * 64);
                float4 reg_b = *(reinterpret_cast<const float4*>(b_block) + tidx + i * 64);
                
                reg_b.w += alpha * reg_a.w;
                reg_b.x += alpha * reg_a.x;
                reg_b.y += alpha * reg_a.y;
                reg_b.z += alpha * reg_a.z;

                *(reinterpret_cast<float4*>(c_block) + tidx + i * 64) = reg_b;
            } else if(bidx + tidx + i * 256 < size) { // we are partially in bounds

                // perform each individually

                float reg_a;
                float reg_b;

                for(int j = 0; j < 4; ++j) {
                    if(j + tidx + i * 256 < size) {
                        reg_a = a_block[j + tidx + i * 256];
                        reg_b = b_block[j + tidx + i * 256];
                        
                        reg_b += alpha * reg_a;

                        c_block[j + tidx + i * 256] = reg_b;
                    }
                }
            }

        }
    }
}


